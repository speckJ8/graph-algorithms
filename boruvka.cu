/**
 *
 * Parellelized version of Boruvka's Algorithm
 * (computation of the Minimum Spanning Tree) using CUDA.
 *
 * */



#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime_api.h>


/**
 * Error checking function adapted from:
 * https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
 */
#define check_cuda(ans) { _check((ans), __FILE__, __LINE__); }
inline hipError_t _check (hipError_t result, const char *file, int line) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                cudaGetErrorString(result), __FILE__, __LINE__);
        assert(result == cudaSuccess);
    }
#endif
    return result;
}


/**
 * Data structure to keep track of a graph vertex.
 *
 * Besides keeping data about the vertex, this data structure is the element
 * of the linked list used to keep track of
 * the connected components during the execution of Boruvka's algorithm.
 * We use a linked list to manage the components so that to merge components,
 * represented by vertices `a` and `b`, we only need to set `a->next = b`
 * without having to make copies.
 */
struct BoruvkaVertex {
    // the label of the vertex
    int label;
    // list of the labels of adjacent vertices
    BoruvkaVertex** adjacent_vertices;
    // number of adjacent vertices
    size_t adjacent_vertices_count;
    // weights of the arrows incident to this vertex
    // `adjacent_vertex_weights[i]` is the weight of the edge
    // connecting this vertex to the vertex `adjacent_vertices[i]`
    int* adjacent_vertex_weights;
    // pointer to the head of the component list where thisvertex is
    // currently
    BoruvkaVertex* component_head;
    // next vertex in the component where this vertex is currently
    BoruvkaVertex* component_next;
    // indicates whether or not this vertex is the head of some
    // component
    bool is_head;
};

typedef BoruvkaVertex** BoruvkaGraph;


/**
 * Find the edge of minimal cost that is incident to a component.
 * 
 * @param graph                the adjacency list of the graph
 * @param components           the list of current components
 * @param component_count      the number of components
 *
 * @return the edge found for `component[c]` will have its source in
           `found_source[c]` and its target in `found_target[c]`
 */
__global__
void boruvka_find_min_edges (BoruvkaVertex** component_heads,
                             BoruvkaVertex** found_source,
                             BoruvkaVertex** found_target,
                             size_t component_count) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int c = index; c < component_count; c += stride) {
        BoruvkaVertex* curr = component_heads[c];
        BoruvkaVertex* edge_source = NULL;
        BoruvkaVertex* edge_target = NULL;
        int edge_weight = -1;
        // find a vertex that is in a different component that has
        // the minimum weight possible
        while (curr != NULL) {
            for (int i = 0; i < curr->adjacent_vertices_count; i++) {
                BoruvkaVertex* adjacent_vertex = curr->adjacent_vertices[i];
                if (adjacent_vertex->component_head != curr->component_head &&
                    (curr->adjacent_vertex_weights[i] < edge_weight ||
                     edge_weight == -1)) {
                    edge_source = curr;
                    edge_target = adjacent_vertex;
                    edge_weight = curr->adjacent_vertex_weights[i];
                }
            }
            // next vertex in the component
            curr = curr->component_next;
        }
        if (edge_source != NULL && edge_target != NULL) {
            found_source[c] = edge_source;
            found_target[c] = edge_target;
        }
    }
}


size_t boruvka_merge_components (BoruvkaVertex** found_source,
                                 BoruvkaVertex** found_target,
                                 size_t component_count) {
    size_t _component_count = component_count;
    for (int c = 0; c < component_count; c++) {
        // source and target vertices of the edge found for
        // the current component
        BoruvkaVertex* source = found_source[c];
        BoruvkaVertex* target= found_target[c];

        if (target->component_head == source->component_head) {
            continue;
        }

        printf("-- new edge (%d, %d)\n", source->label, target->label);

        if (source != NULL && target != NULL) {
            BoruvkaVertex* curr = source;
            while (curr->component_next != NULL) {
                curr = curr->component_next;
            }
            // merge the component containing `source` with the component
            // containing `target` by making the last vertex in the former
            // component point to the first vertex in the latter
            curr->component_next = target->component_head;

            // now update the vertices in the component of `target` by
            // setting their head to be the head of the new merged component
            curr = target->component_head;
            target->component_head->is_head = false;
            while (curr != NULL) {
                curr->component_head = source->component_head;
                curr = curr->component_next;
            }
            _component_count -= 1;
        }
    }
    return _component_count;
}


void boruvka (BoruvkaGraph graph, size_t vertex_count) {

    size_t component_count = vertex_count;
    int threads = 1024;

    BoruvkaVertex** component_heads;
    BoruvkaVertex** found_sources;
    BoruvkaVertex** found_targets;
    hipMallocManaged(&component_heads, component_count*sizeof(BoruvkaVertex*));
    hipMallocManaged(&found_sources, component_count*sizeof(BoruvkaVertex*));
    hipMallocManaged(&found_targets, component_count*sizeof(BoruvkaVertex*));

    // start with `vertex_count` components, each being a trivial
    // tree with one vertex and no edges
    for (int v = 0; v < vertex_count; v++) {
        component_heads[v] = graph[v];
        component_heads[v]->component_head = component_heads[v];
        component_heads[v]->component_next = NULL;
        component_heads[v]->is_head = true;
    }

    while (component_count != 1) {
        int blocks = (component_count + threads - 1) / threads;
        boruvka_find_min_edges<<<threads, blocks>>>(
                component_heads,
                found_sources,
                found_targets,
                component_count
        );
        check_cuda( hipDeviceSynchronize() );

        size_t new_component_count = boruvka_merge_components(
                found_sources,
                found_targets,
                component_count
        );

        BoruvkaVertex** new_component_heads;
        hipMallocManaged(
                &new_component_heads,
                new_component_count*sizeof(BoruvkaVertex*)
        );

        int nc = 0;
        for (int c = 0; c < component_count; c++) {
            if (component_heads[c]->is_head) {
                new_component_heads[nc++] = component_heads[c];
            }
        }

        hipFree(component_heads);
        component_heads = new_component_heads;
        component_count = new_component_count;
    }

    hipFree(component_heads);
    hipFree(found_sources);
    hipFree(found_targets);
}


int main (void) {
    size_t SIZE = 4;
    BoruvkaVertex *a, *b, *c, *d;
    hipMallocManaged(&a, sizeof(BoruvkaVertex));
    hipMallocManaged(&b, sizeof(BoruvkaVertex));
    hipMallocManaged(&c, sizeof(BoruvkaVertex));
    hipMallocManaged(&d, sizeof(BoruvkaVertex));

    /**
     * Initialize the simple graph:
     *        1
     *    0 ----- 1
     *    |       |
     *  3 |       | 2
     *    |       |
     *    3 ----- 2
     *        1
     * */

    a->label = 0;
    a->adjacent_vertices_count = 2;
    hipMallocManaged(
            &a->adjacent_vertices,
            a->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    hipMallocManaged(
            &a->adjacent_vertex_weights,
            a->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    a->adjacent_vertices[0] = b;
    a->adjacent_vertex_weights[0] = 1;
    a->adjacent_vertices[1] = d;
    a->adjacent_vertex_weights[1] = 3;

    b->label = 1;
    b->adjacent_vertices_count = 2;
    hipMallocManaged(
            &b->adjacent_vertices,
            b->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    hipMallocManaged(
            &b->adjacent_vertex_weights,
            b->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    b->adjacent_vertices[0] = a;
    b->adjacent_vertex_weights[0] = 1;
    b->adjacent_vertices[1] = c;
    b->adjacent_vertex_weights[1] = 2;

    c->label = 2;
    c->adjacent_vertices_count = 2;
    hipMallocManaged(
            &c->adjacent_vertices,
            c->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    hipMallocManaged(
            &c->adjacent_vertex_weights,
            c->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    c->adjacent_vertices[0] = b;
    c->adjacent_vertex_weights[0] = 2;
    c->adjacent_vertices[1] = d;
    c->adjacent_vertex_weights[1] = 1;

    d->label = 3;
    d->adjacent_vertices_count = 2;
    hipMallocManaged(
            &d->adjacent_vertices,
            d->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    hipMallocManaged(
            &d->adjacent_vertex_weights,
            d->adjacent_vertices_count*sizeof(BoruvkaVertex*)
    );
    d->adjacent_vertices[0] = c;
    d->adjacent_vertex_weights[0] = 1;
    d->adjacent_vertices[1] = a;
    d->adjacent_vertex_weights[1] = 3;

    BoruvkaGraph graph;
    hipMallocManaged(&graph, SIZE*sizeof(BoruvkaVertex*));
    graph[0] = a;
    graph[1] = b;
    graph[2] = c;
    graph[3] = d;

    boruvka(graph, SIZE);

    hipFree(a->adjacent_vertices);
    hipFree(a->adjacent_vertex_weights);
    hipFree(a);

    hipFree(b->adjacent_vertices);
    hipFree(b->adjacent_vertex_weights);
    hipFree(b);

    hipFree(c->adjacent_vertices);
    hipFree(c->adjacent_vertex_weights);
    hipFree(c);

    hipFree(d->adjacent_vertices);
    hipFree(d->adjacent_vertex_weights);
    hipFree(d);

    hipFree(graph);

    return 0;
}
